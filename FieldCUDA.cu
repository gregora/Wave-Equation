#include "hip/hip_runtime.h"
#include "Field.h"

#include "hip/hip_runtime.h"
#include ""

Field::Field(uint width, uint height, float dx){
    this->width = width;
    this->height = height;
    this->particles = new Particle[width*height];
}

Field::~Field(){
    delete[] particles;
}

Particle* Field::get(uint x, uint y) const {
    return &particles[y*width + x];
}

__global__
void physics_CUDA(float dt, float dx, Particle* particles, Particle* new_particles, uint width){

    uint x = blockIdx.x + 1;
    uint y = threadIdx.x + 1;


    Particle& p = particles[y*width + x];
    Particle& p_new = new_particles[y*width + x];

    Particle& pL = particles[y*width + x - 1];
    Particle& pR = particles[y*width + x + 1];
    Particle& pU = particles[(y - 1)*width + x];
    Particle& pD = particles[(y + 1)*width + x];


    float alpha = p.c * dt / dx;
    float u_new = -(p.u - dt*p.du) + 2*p.u + alpha*alpha*(pL.u + pR.u + pU.u + pD.u - 4*p.u);

    p_new.c = p.c;
    p_new.du = (u_new - p.u) / dt;
    p_new.u = u_new * pow(0.995, dt);


}

void Field::physics(float dt, uint substeps){

    dt /= substeps;

    Particle* device_particles1;
    Particle* device_particles2;

    hipMalloc(&device_particles1, width*height*sizeof(Particle));
    hipMalloc(&device_particles2, width*height*sizeof(Particle));

    hipMemcpy(device_particles1, particles, width*height*sizeof(Particle), hipMemcpyHostToDevice);

    for(uint i = 0; i < substeps; i++){

        physics_CUDA<<<width - 2, height - 2>>>(dt, dx, device_particles1, device_particles2, width);
        hipDeviceSynchronize();

        Particle *temp = device_particles1;
        device_particles1 = device_particles2;
        device_particles2 = temp;

    }
    hipMemcpy(particles, device_particles1, width*height*sizeof(Particle), hipMemcpyDeviceToHost);

    hipFree(device_particles1);
    hipFree(device_particles2);

}

void Field::draw(sf::RenderTarget& target, sf::RenderStates states) const {
    
    sf::RectangleShape rect(sf::Vector2f(scale, scale));

    for(uint y = 0; y < height; y++){
        for(uint x = 0; x < width; x++){
            const Particle& p = *get(x, y);

            float u = p.u;

            if(u < -0.5f) u = -0.5f;
            if(u > 0.5f) u = 0.5f;

            u = u + 0.5f;

            rect.setPosition(x*scale, y*scale);
            rect.setFillColor(sf::Color(255*u, 255*u, 255*u));

            target.draw(rect);
        }
    }
}